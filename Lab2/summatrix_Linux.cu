#include "hip/hip_runtime.h"
// Hands-on Lab 2 for linux 
// Compile: nvcc ... -I \usr\etc summatrix.cu -o summ
// NB: check your own path for the common\inc directory to include as -I 
// Run: ./summ

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime_api.h>

/*
 * This example implements matrix element-wise addition on the host and GPU.
 * sumMatrixOnHost iterates over the rows and columns of each matrix, adding
 * elements from A and B together and storing the results in C. The current
 * offset in each matrix is stored using pointer arithmetic. sumMatrixOnGPU2D
 * implements the same logic, but using CUDA threads to process each matrix.
 */

void initialData(float *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 10.0f;
    }
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            printf("host %f gpu %f ", hostRef[i], gpuRef[i]);
            printf("Arrays do not match.\n\n");
            break;
        }
    }
}

// grid 2D block 2D
__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    // unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    // unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int psBlockIdxX = blockIdx.x % (NX / blockDim.x); //pseudo blockIdx.x
    unsigned int psBlockIdxY = blockIdx.x / (NX / blockDim.x); //pseudo blockIdx.y
    //now calculate as before with pseudo indexes
    unsigned int ix = psBlockIdxX * blockDim.x + threadIdx.x;
    unsigned int iy = psBlockIdxY * blockDim.y + threadIdx.y;
    unsigned int idx = iy * NX + ix;

    

    if (ix < NX && iy < NY)
    {
        C[idx] = A[idx] + B[idx];
    }
}

dim3 calculateGridSize(int nx, int ny, int blockX, int blockY)
{
    dim3 grid( ((nx + blockX - 1) / blockX) * ((ny + blockY - 1) / blockY) );
    float Fgridx = ((nx + blockX - 1) / blockX);
    float Fgridy = ((ny + blockY - 1) / blockY);
    float Fgrid = Fgridx * Fgridy;
    int Igrid = ((nx + blockX - 1) / blockX) * ((ny + blockY - 1) / blockY);
    if (Fgrid > Igrid)
    {
        dim3 grid = (Igrid + 1);
    }


    return grid; 
}



int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    checkCudaErrors(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 12;  // 14
    int ny = 1 << 12;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    initialData(h_A, nxy);
    initialData(h_B, nxy);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    sumMatrixOnHost (h_A, h_B, hostRef, nx, ny);

	// malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    checkCudaErrors(hipMalloc((void **)&d_MatA, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_MatB, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_MatC, nBytes));

   // setup kernel launch parameters
    int dimx = 64; // default block size if no runtime parameters given
    int dimy = 2;

    if(argc > 2)
    {
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }
	
	dim3 block(dimx, dimy);
    //dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    //adjusted to keep same number of total threads
    //dim3 grid( ((nx + block.x - 1) / block.x) * ((ny + block.y - 1) / block.y) ); 
    //dim3 grid( (nx * ny) / (block.x * block.y));
    dim3 grid = calculateGridSize(nx, ny, block.x, block.y);

    // transfer data from host to device
    checkCudaErrors(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

	// initialise CUDA timing
	float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);  // start timing

	// execute the kernel
    checkCudaErrors(hipDeviceSynchronize());
    sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    hipEventRecord(stop);
	checkCudaErrors(hipEventSynchronize(stop));
	hipEventElapsedTime(&milli, start, stop);  // time random generation

	printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> (ms): %f \n", grid.x, grid.y,
           block.x, block.y, milli);

    checkCudaErrors(hipGetLastError());

    // copy kernel result back to host side
    checkCudaErrors(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    // checkCudaErrors device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    checkCudaErrors(hipFree(d_MatA));
    checkCudaErrors(hipFree(d_MatB));
    checkCudaErrors(hipFree(d_MatC));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    checkCudaErrors(hipDeviceReset());

    return EXIT_SUCCESS;
}
