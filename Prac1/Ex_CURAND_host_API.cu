#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

/*  A Host API example program using the CURAND library
    which generates 100 pseudorandom floats,
    taken from the cuda toolkit documentation. 

    compile with: nvcc Ex_CURAND_host_API.cu -lcurand -o ExhostAPI
*/

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

int main(int argc, char *argv[])
{
    size_t n = 100;
    size_t i;
    hiprandGenerator_t gen;
    float *devData, *hostData;

    /* Allocate n floats on host */
    hostData = (float *)calloc(n, sizeof(float));

    /* Allocate n floats on device */
    CUDA_CALL(hipMalloc((void **)&devData, n*sizeof(float)));

    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT));

    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,
                1234ULL));

    /* Generate n floats on device */
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
        hipMemcpyDeviceToHost));

    /* Show result */
    for(i = 0; i < n; i++) {
        printf("%1.4f ", hostData[i]);
    }
    printf("\n");

    /* Cleanup */
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(devData));
    free(hostData);
    return EXIT_SUCCESS;
}