#include "hip/hip_runtime.h"
/*  Program to compute Pi using Monte Carlo methods, on the GPU
    Compile with nvcc PMCPi.cu -lcurand -o PMCPi
    Run with PMCPi
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>


__global__ void generate_kernel(int *result, int calcpt)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x; 
    int count = 0;
    double x, y, z;

    //create state in same kernel
    hiprandState state;
    hiprand_init(1234, id, 0, &state);

    //generate pseudo-random unsigned int
    for (int n = 0; n < calcpt; n++) {
        /*  hiprand uniform returns pseudo random numbers in the range 0 to 1
            the algorithm still works because we are just calculating for a 
            qurter of a circle
        */
        x = hiprand_uniform_double(&state);
        y = hiprand_uniform_double(&state);
        z = x*x + y*y;
        if (z<=1) count++;
    }

    //Store results
    result[id] += count;
}

int main(int argc, char **argv)
{
    double pi;
    int *devResults, *hostResults;
    int count = 0;
    int nt = 2048 * 6; //total number of threads (6 SMs * 2048 threads)
    int niter, calcpt, riter;
    //Take input from command line
    if (argc != 2)
    {
      printf("Usage: PMCPi <number_of_iterations>\n");
      exit(EXIT_FAILURE);
    } 
    niter = atoi(argv[1]);
    if (niter <= 0) 
    {
      printf("Number of iterations must be a positive integer\n");
      exit(EXIT_FAILURE);
    }
    calcpt = niter / nt; //calculations per thread
    riter = nt * calcpt; //rounded iterations

    //Allocate space for results on host
    hostResults = (int *)calloc(nt, sizeof(int));

    //Allocate space for results on device 
    checkCudaErrors(hipMalloc((void **)&devResults,
                    nt * sizeof(int)));

    
    /*  Setup prng states
        2048 threads per SM = 32 blocks, 64 threads each 
        32 blocks * 6 SMs = 192 blocks total 
        (good place to return for optimisation)
    */
    
    //Generate pseudo-random
    generate_kernel<<<192, 64>>>(devResults, calcpt);

    //Copy device memory to host
    checkCudaErrors(hipMemcpy(hostResults, devResults,
                    nt * sizeof(int), hipMemcpyDeviceToHost));

    //Calculate total count
    for (int i = 0; i < nt; i++) {
        count += hostResults[i];
    }
    //calculate pi
    pi = (double)count/riter*4; 
    printf("# of trials= %d , estimate of pi is %g \n",riter,pi);
    
    //clean up
    free(hostResults);
    checkCudaErrors(hipFree(devResults));
    checkCudaErrors(hipDeviceReset());
    
    return 0;

}