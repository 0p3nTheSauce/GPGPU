#include "hip/hip_runtime.h"
/*************************************************
 * Laplace Serial C Version
 *
 * Temperature is initially 0.0
 * Boundaries are as follows:
 *
 *      0         T         0
 *   0  +-------------------+  0
 *      |                   |
 *      |                   |
 *      |                   |
 *   T  |                   |  T
 *      |                   |
 *      |                   |
 *      |                   |
 *   0  +-------------------+ 100
 *      0         T        100
 *
 *  Copyright John Urbanic, PSC 2017
 *
 ************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#define COLUMNS    254
#define ROWS       190

#ifndef MAX_ITER
#define MAX_ITER 1
#endif

// largest permitted change in temp (This value takes about 3400 steps)
#define MAX_TEMP_ERROR 0.01

double Temperature[ROWS+2][COLUMNS+2];      // temperature grid
double Temperature_last[ROWS+2][COLUMNS+2]; // temperature grid from last iteration
//Added by me
double Temp_Temperature[ROWS+2][COLUMNS+2]; //used to temporarily store the values of Temperature


//   helper routines
void initialize();
void track_progress(int iter);
// Added by me
void printMatrix(double *matrix, int rows, int cols);
void printMatrixSubset(double *matrix, int rows, int cols,
                     int fromRow, int toRow,int fromCol, int toCol);
void setTo(double *matrix, int rows, int cols, double val);
void setToInc(double *matrix, int rows, int cols);
void laplace(double *dt, int *iteration);
int checkResult();
//Kernel prototypes
__global__ void avn_tmpchng(double *Temp, double *Temp_last, int rows, int cols, double *dts,
                            int workPT);

int main(int argc, char *argv[]) {

    // int i, j;                                            // grid indexes
    int max_iterations;                                  // number of iterations
    int iteration=1;                                     // current iteration
    double dt=100;                                       // largest change in t
    struct timeval start_time, stop_time, elapsed_time;  // timers

    max_iterations = MAX_ITER;

    gettimeofday(&start_time,NULL); // Unix timer

    //malloc device
    double *d_Temp, *d_Temp_last, *d_dts;
    int rows = ROWS+2, cols = COLUMNS+2;
    int nelems = rows*cols;
    int nBytes = nelems*sizeof(double);
    checkCudaErrors(hipMalloc((void **)&d_Temp, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_Temp_last, nBytes));
    //dts could be ROWS * COLS but for simpliity 1-to-1 correspondence with Temp and Temp_last
    checkCudaErrors(hipMalloc((void **)&d_dts, nBytes)); 
    //malloc host dts
    double *h_dts;
    h_dts = (double *)malloc(nBytes);

    //for printing 
    int fromRow = 0;
    int toRow = 192;
    int fromCol = 234;
    int toCol = 256;
    initialize();                   // initialize Temp_last including boundary conditions
    setTo(*Temperature, rows, cols, 0.0);
    printf("Temperature after initialization: ");
    printMatrixSubset(*Temperature, rows, cols, fromRow, toRow, fromCol, toCol);
    // printf("Temperature_last after initialization: ");
    // printMatrixSubset(*Temperature_last, rows, cols, fromRow, toRow, fromCol, toCol);
    //Transfer data from host to device
    checkCudaErrors(hipMemcpy(d_Temp, Temperature, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Temp_last, Temperature_last, nBytes, hipMemcpyHostToDevice));
    
    //setup kernel
    dim3 block(1024); //for testing, to change later
    dim3 grid(12);
    int workPT = (rows * cols) / 12288;

    //test if kernel working 
    //max_iterations = 1;

    //do until error is minimal or until max steps
    while ( dt > MAX_TEMP_ERROR && iteration <= max_iterations ) {


        checkCudaErrors(hipDeviceSynchronize());
        // main calculation: average my four neighbors    
        avn_tmpchng<<<grid, block>>>(d_Temp, d_Temp_last, rows, cols, d_dts, workPT);
        checkCudaErrors(hipGetLastError());
        
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipMemcpy(Temperature, d_Temp, nBytes, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(Temperature_last, d_Temp_last, nBytes, hipMemcpyDeviceToHost));
        // printf("Temperature after kernel: ");
        // printMatrixSubset(*Temperature, rows, cols, fromRow, toRow, fromCol, toCol);
        // printf("Temperature_last after kernel: ");
        // printMatrixSubset(*Temperature_last, rows, cols, fromRow, toRow, fromCol, toCol);

        dt = 0.0; // reset largest temperature change
        //checkCudaErrors(hipMemset(d_dts, 0, nBytes));
        
        checkCudaErrors(hipDeviceSynchronize());
        //copy dts to host
        checkCudaErrors(hipMemcpy(h_dts, d_dts, nBytes, hipMemcpyDeviceToHost));
        //find dt
        checkCudaErrors(hipDeviceSynchronize());
        for (int i = 0; i < nBytes; i++) {
            dt = fmax(h_dts[i], dt);
        }


        //periodically print test values
        if((iteration % 100) == 0) {
            checkCudaErrors(hipMemcpy(Temperature, d_Temp, nBytes, hipMemcpyDeviceToHost));
 	        track_progress(iteration); 
        }

	iteration++;
    }
    //laplace(&dt, &iteration);

    gettimeofday(&stop_time,NULL);
    //copy results back to host
    checkCudaErrors(hipMemcpy(Temperature, d_Temp, nBytes, hipMemcpyDeviceToHost));
    // checkCudaErrors(hipMemcpy(Temperature_last, d_Temp_last, nBytes, hipMemcpyDeviceToHost));
    printf("Temperature after laplace: ");
    printMatrixSubset(*Temperature, rows, cols, fromRow, toRow, fromCol, toCol);
    // printf("Temperature_last after laplace: ");
    // printMatrixSubset(*Temperature_last, rows, cols, fromRow, toRow, fromCol, toCol);
	timersub(&stop_time, &start_time, &elapsed_time); // Unix time subtract routine

    printf("\nMax error at iteration %d was %f\n", iteration-1, dt);
    printf("Total time was %f seconds.\n", elapsed_time.tv_sec+elapsed_time.tv_usec/1000000.0);

    printf("Check result: \n");
    if (checkResult()){
        printf("Results correct\n");
    } else {
        printf("Results incorrect\n");
    }

    //Deallocate memory
    checkCudaErrors(hipFree(d_Temp));
    checkCudaErrors(hipFree(d_Temp_last));
    checkCudaErrors(hipFree(d_dts));
    free(h_dts);
    //reset device
    checkCudaErrors(hipDeviceReset());

    exit(0);
}

__global__ void avn_tmpchng(double *Temp, double *Temp_last, int rows, int cols,
                            double *dts, int workPT)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = iy * cols + ix;
    double dt = 0;
    
    int startIdx = tid * workPT;
    for (int idx = startIdx; idx < startIdx + workPT; idx++){
        if (idx > cols && idx < (cols * (rows - 1)) && idx % cols != 0 && (idx+1) % cols != 0 ) {
        //     Temp[idx] = 0.25 * (Temp_last[idx+1] + Temp_last[idx-1] +
        //                             Temp_last[idx+cols] + Temp_last[idx-cols]);
        //     dt = fmax(fabs(Temp[idx] - Temp_last[idx]), dt);
        //     Temp_last[idx] = Temp[idx];
               Temp[idx] = 1.11;
        }
        if (idx < cols * rows){
            dts[idx] = dt;
        }
        
    }
}


//laplace algorithm as a function
void laplace(double *dt, int *iteration) {
    //create local variables
    int max_iterations = MAX_ITER;
    double local_dt = *dt;
    int local_iteration = *iteration;
    int i, j;
    //laplace algorithm
    // do until error is minimal or until max steps
    while ( local_dt > MAX_TEMP_ERROR && local_iteration <= max_iterations ) {

        // main calculation: average my four neighbors    
        for(i = 1; i <= ROWS; i++) {
            for(j = 1; j <= COLUMNS; j++) {
                Temperature[i][j] = 0.25 * (Temperature_last[i+1][j] + Temperature_last[i-1][j] +
                                            Temperature_last[i][j+1] + Temperature_last[i][j-1]);
            }
        }
        
        local_dt = 0.0; // reset largest temperature change

        // copy grid to old grid for next iteration and find latest dt
        for(i = 1; i <= ROWS; i++){
            for(j = 1; j <= COLUMNS; j++){
	      local_dt = fmax( fabs(Temperature[i][j]-Temperature_last[i][j]), local_dt);
	      Temperature_last[i][j] = Temperature[i][j];
            }
        }
        
        // periodically print test values
        if((local_iteration % 100) == 0) {
 	    track_progress(local_iteration);
        }

	local_iteration++;
    }
    //pass dt and iterations back to main
    *dt = local_dt;
    *iteration = local_iteration;
}

//check that the output is correct
int checkResult(){
    int iteration=1;                                     // current iteration
    double dt=100;
    int rows = ROWS+2;
    int cols = COLUMNS+2;
    int nBytes = (rows) * (cols) * sizeof(double);
    int i, j;
    const double maxErr = 1e-9; // maximum error for floating point comparison
    memcpy(Temp_Temperature, Temperature, nBytes);
    initialize();
    laplace(&dt, &iteration);
    // printMatrix(*Temp_Temperature, ROWS+2, COLUMNS+2);
    for (i = 0; i < rows; i++){
        for (j = 0; j < cols; j++){
            if (abs(Temp_Temperature[i][j] - Temperature[i][j]) > maxErr){
                printf("Temp_Temperature[%d][%d]: %g\n", i, j, Temp_Temperature[i][j]);
                printf("Temperature[%d][%d]: %g\n", i, j, Temperature[i][j]);
                return 0;
            } 
        }
    }
    return 1;
}

// Print the matrix
void printMatrix(double *matrix, int rows, int cols) {
    printf("Matrix:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%7.2f ", *(matrix + i * cols + j));
        }
        printf("\n");
    }
}

// Print a subset of the matrix
void printMatrixSubset(double *matrix, int rows, int cols,
                     int fromRow, int toRow,int fromCol, int toCol) {
    printf("Matrix:\n");
    for (int i = fromRow; i < toRow; i++) {
        for (int j = fromCol; j < toCol; j++) {
            printf("%7.2f ", *(matrix + i * cols + j));
            //printf("%d ", matrix[i][j]);
        }
        printf("\n");
    }
}


//set all values of a matrix to same values
void setTo(double *matrix, int rows, int cols, double val) {
    for (int i = 0; i < rows * cols; i++) {
        *(matrix + i) = val;
    }
}
//set all values of matrix to incrementing valeus
void setToInc(double *matrix, int rows, int cols) {
    int i, j;
    int val = 0;
    for (i = 0; i < rows; i++){
        for(j=0; j < cols; j++){
            Temperature_last[i][j] = val;
            val++;
        }
    }
}

// initialize plate and boundary conditions
// Temp_last is used to to start first iteration
void initialize(){

    int i,j;

    for(i = 0; i <= ROWS+1; i++){
        for (j = 0; j <= COLUMNS+1; j++){
            Temperature_last[i][j] = 0.0;
        }
    }

    // these boundary conditions never change throughout run

    // set left side to 0 and right to a linear increase
    for(i = 0; i <= ROWS+1; i++) {
        Temperature_last[i][0] = 0.0;
        Temperature_last[i][COLUMNS+1] = (100.0/ROWS)*i;
    }
    
    // set top to 0 and bottom to linear increase
    for(j = 0; j <= COLUMNS+1; j++) {
        Temperature_last[0][j] = 0.0;
        Temperature_last[ROWS+1][j] = (100.0/COLUMNS)*j;
    }
}


// print diagonal in bottom right corner where most action is
void track_progress(int iteration) {

    int i;

    printf("---------- Iteration number: %d ------------\n", iteration);
    for(i = ROWS-5; i <= ROWS; i++) {
        printf("[%d,%d]: %5.2f  ", i, i, Temperature[i][i]);
    }
    printf("\n");
}

