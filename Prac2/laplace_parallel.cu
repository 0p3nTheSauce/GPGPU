#include "hip/hip_runtime.h"
/*************************************************
 * Laplace Serial C Version
 *
 * Temperature is initially 0.0
 * Boundaries are as follows:
 *
 *      0         T         0
 *   0  +-------------------+  0
 *      |                   |
 *      |                   |
 *      |                   |
 *   T  |                   |  T
 *      |                   |
 *      |                   |
 *      |                   |
 *   0  +-------------------+ 100
 *      0         T        100
 *
 *  Copyright John Urbanic, PSC 2017
 *
 ************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#define COLUMNS    30//126//10
#define ROWS       30//94//10

#ifndef MAX_ITER
#define MAX_ITER 100
#endif

// largest permitted change in temp (This value takes about 3400 steps)
#define MAX_TEMP_ERROR 0.01

double Temperature[ROWS+2][COLUMNS+2];      // temperature grid
double Temperature_last[ROWS+2][COLUMNS+2]; // temperature grid from last iteration
//Added by me
double Temp_Temperature[ROWS+2][COLUMNS+2]; //used to temporarily store the values of Temperature


//   helper routines
void initialize();
void track_progress(int iter);
// Added by me
void printMatrix(double *matrix, int rows, int cols);
void printMatrixSubset(double *matrix, int rows, int cols,
                     int fromRow, int toRow,int fromCol, int toCol);
void setTo(double *matrix, int rows, int cols, double val);
void setToInc(double *matrix, int rows, int cols);
void laplace(double *dt, int *iteration);
int checkResult();
//Kernel prototypes
__global__ void avn_tmpchng(double *Temp, double *Temp_last, int rows, int cols, double *dts);

int main(int argc, char *argv[]) {

    // int i, j;                                            // grid indexes
    int max_iterations;                                  // number of iterations
    int iteration=1;                                     // current iteration
    double dt=100;                                       // largest change in t
    struct timeval start_time, stop_time, elapsed_time;  // timers

    max_iterations = MAX_ITER;

    gettimeofday(&start_time,NULL); // Unix timer

    //malloc device
    double *d_Temp, *d_Temp_last, *d_dts;
    int rows = ROWS+2, cols = COLUMNS+2;
    int nelems = rows*cols;
    int nBytes = nelems*sizeof(double);
    checkCudaErrors(hipMalloc((void **)&d_Temp, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_Temp_last, nBytes));
    //dts could be ROWS * COLS but for simpliity 1-to-1 correspondence with Temp and Temp_last
    checkCudaErrors(hipMalloc((void **)&d_dts, nBytes)); 
    //malloc host dts
    double *h_dts;
    h_dts = (double *)malloc(nBytes);

    //for printing 
    int fromRow = 22;//86;
    int toRow = 32;//96;
    int fromCol = 22;//118;
    int toCol = 32;//128;
    initialize();                   // initialize Temp_last including boundary conditions
    printf("Temperature after initialization: ");
    printMatrixSubset(*Temperature, rows, cols, fromRow, toRow, fromCol, toCol);
    //setTo(*Temperature_last, rows, cols, 1.0);
    // printf("Temperature_last after initialization: ");
    // printMatrixSubset(*Temperature_last, rows, cols, fromRow, toRow, fromCol, toCol);
    //Transfer data from host to device
    checkCudaErrors(hipMemcpy(d_Temp, Temperature, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Temp_last, Temperature_last, nBytes, hipMemcpyHostToDevice));
    
    //setup kernel
    dim3 block(32, 32); //for testing, to change later
    dim3 grid(3, 4);

    //test if kernel working 
    //max_iterations = 1;

    // do until error is minimal or until max steps
    while ( dt > MAX_TEMP_ERROR && iteration <= max_iterations ) {


        checkCudaErrors(hipDeviceSynchronize());
        // main calculation: average my four neighbors    
        avn_tmpchng<<<grid, block>>>(d_Temp, d_Temp_last, rows, cols, d_dts);
        checkCudaErrors(hipGetLastError());
        
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipMemcpy(Temperature, d_Temp, nBytes, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(Temperature_last, d_Temp_last, nBytes, hipMemcpyDeviceToHost));
        // printf("Temperature after kernel: ");
        // printMatrixSubset(*Temperature, rows, cols, fromRow, toRow, fromCol, toCol);
        // printf("Temperature_last after kernel: ");
        // printMatrixSubset(*Temperature_last, rows, cols, fromRow, toRow, fromCol, toCol);

        dt = 0.0; // reset largest temperature change
        //checkCudaErrors(hipMemset(d_dts, 0, nBytes));
        
        checkCudaErrors(hipDeviceSynchronize());
        //copy dts to host
        checkCudaErrors(hipMemcpy(h_dts, d_dts, nBytes, hipMemcpyDeviceToHost));
        //find dt
        checkCudaErrors(hipDeviceSynchronize());
        for (int i = 0; i < nBytes; i++) {
            dt = fmax(h_dts[i], dt);
        }


        //periodically print test values
        if((iteration % 100) == 0) {
            checkCudaErrors(hipMemcpy(Temperature, d_Temp, nBytes, hipMemcpyDeviceToHost));
 	        track_progress(iteration); 
        }

	iteration++;
    }

    //copy results back to host
    checkCudaErrors(hipMemcpy(Temperature, d_Temp, nBytes, hipMemcpyDeviceToHost));
    
    // checkCudaErrors(hipMemcpy(Temperature_last, d_Temp_last, nBytes, hipMemcpyDeviceToHost));
    printf("Temperature after laplace: ");
    printMatrixSubset(*Temperature, rows, cols, fromRow, toRow, fromCol, toCol);
    printf("Check result: \n");
    if (checkResult()){
        printf("Results correct\n");
    } else {
        printf("Results incorrect");
    }
    // printf("Temperature_last after laplace: ");
    // printMatrixSubset(*Temperature_last, rows, cols, fromRow, toRow, fromCol, toCol);
    gettimeofday(&stop_time,NULL);
	timersub(&stop_time, &start_time, &elapsed_time); // Unix time subtract routine

    printf("\nMax error at iteration %d was %f\n", iteration-1, dt);
    printf("Total time was %f seconds.\n", elapsed_time.tv_sec+elapsed_time.tv_usec/1000000.0);

    //Deallocate memory
    checkCudaErrors(hipFree(d_Temp));
    checkCudaErrors(hipFree(d_Temp_last));
    checkCudaErrors(hipFree(d_dts));
    free(h_dts);
    //reset device
    checkCudaErrors(hipDeviceReset());

    exit(0);
}

__global__ void avn_tmpchng(double *Temp, double *Temp_last, int rows, int cols,
                            double *dts)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = iy * cols + ix;
    double dt = 0;
    if (ix > 0 && ix < cols-1 && iy > 0 && iy < rows-1) 
    {
        Temp[idx] = 0.25 * (Temp_last[idx+1] + Temp_last[idx-1] +
                                    Temp_last[idx+cols] + Temp_last[idx-cols]);
        dt = fmax(fabs(Temp[idx] - Temp_last[idx]), dt);
        Temp_last[idx] = Temp[idx];
    }
    dts[idx] = dt;
}

//laplace algorithm as a function
void laplace(double *dt, int *iteration) {
    //create local variables
    int max_iterations = MAX_ITER;
    double local_dt = *dt;
    int local_iteration = *iteration;
    int i, j;
    //laplace algorithm
    // do until error is minimal or until max steps
    while ( local_dt > MAX_TEMP_ERROR && local_iteration <= max_iterations ) {

        // main calculation: average my four neighbors    
        for(i = 1; i <= ROWS; i++) {
            for(j = 1; j <= COLUMNS; j++) {
                Temperature[i][j] = 0.25 * (Temperature_last[i+1][j] + Temperature_last[i-1][j] +
                                            Temperature_last[i][j+1] + Temperature_last[i][j-1]);
            }
        }
        
        local_dt = 0.0; // reset largest temperature change

        // copy grid to old grid for next iteration and find latest dt
        for(i = 1; i <= ROWS; i++){
            for(j = 1; j <= COLUMNS; j++){
	      local_dt = fmax( fabs(Temperature[i][j]-Temperature_last[i][j]), local_dt);
	      Temperature_last[i][j] = Temperature[i][j];
            }
        }
        
        // periodically print test values
        if((local_iteration % 100) == 0) {
 	    track_progress(local_iteration);
        }

	local_iteration++;
    }
    //pass dt and iterations back to main
    *dt = local_dt;
    *iteration = local_iteration;
}

//check that the output is correct
int checkResult(){
    int iteration=1;                                     // current iteration
    double dt=100;
    int nBytes = (ROWS+2) * (COLUMNS+2) * sizeof(double);
    int i, j;
    const double maxErr = 1e-9; // maximum error for floating point comparison
    memcpy(Temp_Temperature, Temperature, nBytes);
    initialize();
    laplace(&dt, &iteration);
    // printMatrix(*Temp_Temperature, ROWS+2, COLUMNS+2);
    for (i = 0; i < ROWS+2; i++){
        for (j = 0; j < COLUMNS+2; j++){
            if (Temp_Temperature[i][j] - Temperature[i][j] > maxErr){
                printf("Temp_Temperature[%d][%d]: %g\n", i, j, Temp_Temperature[i][j]);
                printf("Temperature[%d][%d]: %g\n", i, j, Temperature[i][j]);
                return 0;
            } 

        }
    }
    return 1;
}

// Print the matrix
void printMatrix(double *matrix, int rows, int cols) {
    printf("Matrix:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%7.2f ", *(matrix + i * cols + j));
        }
        printf("\n");
    }
}

// Print a subset of the matrix
void printMatrixSubset(double *matrix, int rows, int cols,
                     int fromRow, int toRow,int fromCol, int toCol) {
    printf("Matrix:\n");
    for (int i = fromRow; i < toRow; i++) {
        for (int j = fromCol; j < toCol; j++) {
            printf("%7.2f ", *(matrix + i * cols + j));
            //printf("%d ", matrix[i][j]);
        }
        printf("\n");
    }
}


//set all values of a matrix to same values
void setTo(double *matrix, int rows, int cols, double val) {
    int i, j;
    for (i = 0; i < rows; i++){
        for(j=0; j < cols; j++){
            Temperature_last[i][j] = val;
        }
    }
}
//set all values of matrix to incrementing valeus
void setToInc(double *matrix, int rows, int cols) {
    int i, j;
    int val = 0;
    for (i = 0; i < rows; i++){
        for(j=0; j < cols; j++){
            Temperature_last[i][j] = val;
            val++;
        }
    }
}

// initialize plate and boundary conditions
// Temp_last is used to to start first iteration
void initialize(){

    int i,j;

    for(i = 0; i <= ROWS+1; i++){
        for (j = 0; j <= COLUMNS+1; j++){
            Temperature_last[i][j] = 0.0;
        }
    }

    // these boundary conditions never change throughout run

    // set left side to 0 and right to a linear increase
    for(i = 0; i <= ROWS+1; i++) {
        Temperature_last[i][0] = 0.0;
        Temperature_last[i][COLUMNS+1] = (100.0/ROWS)*i;
    }
    
    // set top to 0 and bottom to linear increase
    for(j = 0; j <= COLUMNS+1; j++) {
        Temperature_last[0][j] = 0.0;
        Temperature_last[ROWS+1][j] = (100.0/COLUMNS)*j;
    }
}


// print diagonal in bottom right corner where most action is
void track_progress(int iteration) {

    int i;

    printf("---------- Iteration number: %d ------------\n", iteration);
    for(i = ROWS-5; i <= ROWS; i++) {
        printf("[%d,%d]: %5.2f  ", i, i, Temperature[i][i]);
    }
    printf("\n");
}

