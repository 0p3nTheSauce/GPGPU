#include "hip/hip_runtime.h"
/*************************************************
 * Laplace Serial C Version
 *
 * Temperature is initially 0.0
 * Boundaries are as follows:
 *
 *      0         T         0
 *   0  +-------------------+  0
 *      |                   |
 *      |                   |
 *      |                   |
 *   T  |                   |  T
 *      |                   |
 *      |                   |
 *      |                   |
 *   0  +-------------------+ 100
 *      0         T        100
 *
 *  Copyright John Urbanic, PSC 2017
 *
 ************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#define COLUMNS    10
#define ROWS       10

#ifndef MAX_ITER
#define MAX_ITER 100
#endif

// largest permitted change in temp (This value takes about 3400 steps)
#define MAX_TEMP_ERROR 0.01

double Temperature[ROWS+2][COLUMNS+2];      // temperature grid
double Temperature_last[ROWS+2][COLUMNS+2]; // temperature grid from last iteration


//   helper routines
void initialize();
void track_progress(int iter);
// Added by me
void printMatrix(double *matrix, int rows, int cols);
//Kernel prototypes
__global__ void avneighbours(double *Temp, double *Temp_last, int rows, int cols);
__global__ void tempchange(double *Temp, double *Temp_last, int rows, int cols,
                            double *dts);

int main(int argc, char *argv[]) {

    // int i, j;                                            // grid indexes
    int max_iterations;                                  // number of iterations
    int iteration=1;                                     // current iteration
    double dt=100;                                       // largest change in t
    struct timeval start_time, stop_time, elapsed_time;  // timers

    max_iterations = MAX_ITER;

    gettimeofday(&start_time,NULL); // Unix timer

    //malloc device
    double *d_Temp, *d_Temp_last, *d_dts;
    int rows = ROWS+2, cols = COLUMNS+2;
    int nelems = rows*cols;
    int nBytes = nelems*sizeof(double);
    checkCudaErrors(hipMalloc((void **)&d_Temp, nBytes));
    checkCudaErrors(hipMalloc((void **)&d_Temp_last, nBytes));
    //dts could be ROWS * COLS but for simpliity 1-to-1 correspondence with Temp and Temp_last
    checkCudaErrors(hipMalloc((void **)&d_dts, nBytes)); 
    //malloc host dts
    double *h_dts;
    h_dts = (double *)malloc(nBytes);


    initialize();                   // initialize Temp_last including boundary conditions
    printMatrix(*Temperature, rows+2,cols+2 );

    //Transfer data from host to device
    checkCudaErrors(hipMemcpy(d_Temp, Temperature, nBytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Temp_last, Temperature_last, nBytes, hipMemcpyHostToDevice));
    
    //setup kernel
    dim3 block(32); //for testing, to change later
    dim3 grid(1);

    // do until error is minimal or until max steps
    while ( dt > MAX_TEMP_ERROR && iteration <= max_iterations ) {


        checkCudaErrors(hipDeviceSynchronize());
        // main calculation: average my four neighbors    
        avneighbours<<<grid, block>>>(d_Temp, d_Temp_last, rows, cols);
        checkCudaErrors(hipGetLastError());
        
        dt = 0.0; // reset largest temperature change
        checkCudaErrors(hipMemset(d_dts, 0, nBytes));

        checkCudaErrors(hipDeviceSynchronize());
        // copy grid to old grid for next iteration and find latest dt
        tempchange<<<grid, block>>>(d_Temp, d_Temp_last, rows, cols, d_dts);
        checkCudaErrors(hipGetLastError());
        
        checkCudaErrors(hipDeviceSynchronize());
        //copy dts to host
        checkCudaErrors(hipMemcpy(h_dts, d_dts, nBytes, hipMemcpyDeviceToHost));
        //find dt
        for (int i = 0; i < nBytes; i++) {
            dt = fmax(h_dts[i], dt);
        }


        // periodically print test values
        // if((iteration % 100) == 0) {
 	    // track_progress(iteration); *********will need to adjust this at some point
        // }

	iteration++;
    }

    //copy results back to host
    checkCudaErrors(hipMemcpy(Temperature, d_Temp, nBytes, hipMemcpyDeviceToHost));

    printMatrix(*Temperature, ROWS+2,COLUMNS+2 );
    gettimeofday(&stop_time,NULL);
	timersub(&stop_time, &start_time, &elapsed_time); // Unix time subtract routine

    printf("\nMax error at iteration %d was %f\n", iteration-1, dt);
    printf("Total time was %f seconds.\n", elapsed_time.tv_sec+elapsed_time.tv_usec/1000000.0);

    //Deallocate memory
    checkCudaErrors(hipFree(d_Temp));
    checkCudaErrors(hipFree(d_Temp_last));
    checkCudaErrors(hipFree(d_dts));
    free(h_dts);
    //reset device
    checkCudaErrors(hipDeviceReset());

    exit(0);
}

__global__ void avneighbours(double *Temp, double *Temp_last, int rows, int cols)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = iy * cols + ix;

    if (ix > 0 && ix < cols-2 && iy > 0 && rows-2) 
    {
        Temp[idx] = 0.25 * (Temp_last[idx+1] + Temp_last[idx-1] +
                                    Temp_last[idx+cols] + Temp_last[idx-cols]);
    }

}

__global__ void tempchange(double *Temp, double *Temp_last, int rows, int cols,
                            double *dts)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = iy * cols + ix;
    double dt = 0;
    if (ix > 0 && ix < cols-2 && iy > 0 && rows-2) 
    {
        dt = fmax(fabs(Temp[idx] - Temp_last[idx]), dt);
        Temp[idx] = Temp_last[idx];
    }
    dts[idx] = dt;
}

// Function definition to print the matrix
void printMatrix(double *matrix, int rows, int cols) {
    printf("Matrix:\n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", *(matrix + i * cols + j));
        }
        printf("\n");
    }
}

// initialize plate and boundary conditions
// Temp_last is used to to start first iteration
void initialize(){

    int i,j;

    for(i = 0; i <= ROWS+1; i++){
        for (j = 0; j <= COLUMNS+1; j++){
            Temperature_last[i][j] = 0.0;
        }
    }

    // these boundary conditions never change throughout run

    // set left side to 0 and right to a linear increase
    for(i = 0; i <= ROWS+1; i++) {
        Temperature_last[i][0] = 0.0;
        Temperature_last[i][COLUMNS+1] = (100.0/ROWS)*i;
    }
    
    // set top to 0 and bottom to linear increase
    for(j = 0; j <= COLUMNS+1; j++) {
        Temperature_last[0][j] = 0.0;
        Temperature_last[ROWS+1][j] = (100.0/COLUMNS)*j;
    }
}


// print diagonal in bottom right corner where most action is
void track_progress(int iteration) {

    int i;

    printf("---------- Iteration number: %d ------------\n", iteration);
    for(i = ROWS-5; i <= ROWS; i++) {
        printf("[%d,%d]: %5.2f  ", i, i, Temperature[i][i]);
    }
    printf("\n");
}

